#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "fp4.h"

extern "C" __global__
void multi_bit_reverse(Fp* io,
                       const uint32_t nBits,
                       const uint32_t count) {
  uint totIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (totIdx < count) {
    uint32_t rowSize = 1 << nBits;
    uint32_t idx = totIdx & (rowSize - 1);
    uint32_t s = totIdx >> nBits;
    uint32_t ridx = __brev(idx) >> (32 - nBits);
    if (idx < ridx) {
      size_t idx1 = s * rowSize + idx;
      size_t idx2 = s * rowSize + ridx;
      Fp tmp = io[idx1];
      io[idx1] = io[idx2];
      io[idx2] = tmp;
    }
  }
}

extern "C" __global__
void batch_evaluate_any(Fp4* out,
                        const Fp* coeffs,
                        const uint32_t* which,
                        const Fp4* xs,
                        const uint32_t eval_count,
                        const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    Fp4 tot;
    Fp4 cur(1);
    const uint32_t id = which[idx];
    const Fp4& x = xs[idx];
    for (uint32_t i = 0; i < eval_count; i++) {
        tot += cur * coeffs[eval_count * id + i];
        cur *= x;
    }
    out[idx] = tot;
  }
}

extern "C" __global__
void batch_expand(Fp* out,
                  const Fp* in,
                  const uint32_t polyCount,
                  const uint32_t outSize,
                  const uint32_t inSize,
                  const uint32_t expandBits) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < outSize) {
    for (uint32_t i = 0; i < polyCount; i++) {
      out[i * outSize + idx] = in[i * inSize + (idx >> expandBits)];
    }
  }
}

extern "C" __global__
void multi_ntt_fwd_step(Fp* io,
                        const Fp* rou,
                        const uint32_t nBits,
                        const uint32_t sBits,
                        const uint32_t cSize) {
  uint32_t gSize = 1 << (nBits - sBits);
  uint32_t sSize = 1 << (sBits - 1);
  uint32_t nSize = 1 << nBits;
  uint32_t sOff = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t sStep = blockDim.x * gridDim.x;
  uint32_t gOff = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t gStep = blockDim.y * gridDim.y;
  uint32_t cOff = threadIdx.z + blockIdx.z * blockDim.z;
  uint32_t cStep = blockDim.z * gridDim.z;

  // Compute the intial multiplier for the sOff: pow(rou[s], sOff)
  Fp curMul(1);
  uint32_t curRou = sBits;
  uint32_t powX = sOff;
  while (curRou > 0) {
    if (powX & 1) {
      curMul = curMul * rou[curRou];
    }
    powX >>= 1;
    curRou--;
  }
  // Compute the multiplier for each loop around s
  int rouStep = __ffs(sSize / sStep);
  Fp stepMul = rou[rouStep];
  for (uint32_t s = sOff; s < sSize; s += sStep) {
    for (uint32_t g = gOff; g < gSize; g += gStep) {
      for (uint32_t c = cOff; c < cSize; c += cStep) {
        Fp a = io[c * nSize + g * 2 * sSize + s];
        Fp b = io[c * nSize + g * 2 * sSize + s + sSize];
        b *= curMul;
        io[c * nSize + g * 2 * sSize + s] = a + b;
        io[c * nSize + g * 2 * sSize + s + sSize] = a - b;
      }
    }
    curMul *= stepMul;
  }
}

extern "C" __global__
void multi_ntt_rev_step(Fp* io,
                        const Fp* rou,
                        const uint32_t nBits,
                        const uint32_t sBits,
                        const uint32_t cSize) {
  uint32_t gSize = 1 << (nBits - sBits);
  uint32_t sSize = 1 << (sBits - 1);
  uint32_t nSize = 1 << nBits;
  uint32_t sOff = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t sStep = blockDim.x * gridDim.x;
  uint32_t gOff = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t gStep = blockDim.y * gridDim.y;
  uint32_t cOff = threadIdx.z + blockIdx.z * blockDim.z;
  uint32_t cStep = blockDim.z * gridDim.z;

  // Compute the intial multiplier for the sOff: pow(rou[s], sOff)
  Fp curMul(1);
  uint32_t curRou = sBits;
  uint32_t powX = sOff;
  while (curRou > 0) {
    if (powX & 1) {
      curMul = curMul * rou[curRou];
    }
    powX >>= 1;
    curRou--;
  }
  // Compute the multiplier for each loop around s
  int rouStep = __ffs(sSize / sStep);
  Fp stepMul = rou[rouStep];
  for (uint32_t s = sOff; s < sSize; s += sStep) {
    for (uint32_t g = gOff; g < gSize; g += gStep) {
      for (uint32_t c = cOff; c < cSize; c += cStep) {
        Fp a = io[c * nSize + g * 2 * sSize + s];
        Fp b = io[c * nSize + g * 2 * sSize + s + sSize];
        io[c * nSize + g * 2 * sSize + s] = a + b;
        io[c * nSize + g * 2 * sSize + s + sSize] = (a - b) * curMul;
      }
    }
    curMul *= stepMul;
  }
}
